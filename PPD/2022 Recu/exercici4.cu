#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__global__ void first_op(float *b, int N){

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i < N){
        b[i] = 0.3 * (N - i);
    }
}

__global__ void second_op(float *a, float *c, int N){

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i < N){
        c[i] = a[i] * i;
    }
}

__global__ void third_op(float *c, float *d, float *norm, int N){

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i > 0 && i < N-1){
        d[i] = (1/3) * (c[i - 1] + c[i] + c[i + 1]);
        norm += c[i] * c[i];
    }
}

void main (int argc, char *argv[])
{
    int N = 4096; 
    int BLOCK_SIZE = 16; 
    float norm;

    float * a = (float*)malloc(sizeof(float)*N);
    float * b = (float*)malloc(sizeof(float)*N);
    float * c = (float*)malloc(sizeof(float)*N);
    float * d = (float*)malloc(sizeof(float)*N);

    float *d_a, *d_b, *d_c, *d_d, *d_norm;

    hipMalloc((void**)&d_a,N*sizeof(float));
    hipMalloc((void**)&d_b,N*sizeof(float));
    hipMalloc((void**)&d_c,N*sizeof(float));
    hipMalloc((void**)&d_d,N*sizeof(float));
    hipMalloc((void**)&d_norm,sizeof(float));


    initialize(a); /* Consider this function initializes a. Do not parallelize it. */

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_norm, 0, sizeof(float), hipMemcpyHostToDevice);
    int NUM_BLOCKS = (N+BLOCK_SIZE-1)/BLOCK_SIZE;

    first_op<<< NUM_BLOCKS, BLOCK_SIZE >>>(d_b, N)
    second_op<<< NUM_BLOCKS, BLOCK_SIZE >>>(d_a, d_c, N)
    third_op<<< NUM_BLOCKS, BLOCK_SIZE >>>(d_c, d_d, d_norm, N)

    hipMemcpy(&d_d, vald, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&d_norm, norm, N*sizeof(float), hipMemcpyDeviceToHost);


    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    hipFree(d_norm);


    printf("The value of the mid point of d is: %f\n", vald);
    printf("The norm of the interior points of c is: %f\n",norm);

}